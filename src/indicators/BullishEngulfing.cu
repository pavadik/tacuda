#include "hip/hip_runtime.h"
#include <indicators/BullishEngulfing.h>
#include <utils/CandleUtils.h>
#include <utils/CudaUtils.h>

__global__ void bullishEngulfingKernel(const float* __restrict__ open,
                                       const float* __restrict__ high,
                                       const float* __restrict__ low,
                                       const float* __restrict__ close,
                                       float* __restrict__ output,
                                       int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > 0 && idx < size) {
        output[idx] = is_bullish_engulfing(open[idx - 1], close[idx - 1],
                                           open[idx], close[idx]) ? 1.0f : 0.0f;
    }
}

void BullishEngulfing::calculate(const float* open, const float* high, const float* low,
                                 const float* close, float* output, int size) noexcept(false) {
    CUDA_CHECK(hipMemset(output, 0xFF, size * sizeof(float)));
    dim3 block = defaultBlock();
    dim3 grid = defaultGrid(size);
    bullishEngulfingKernel<<<grid, block>>>(open, high, low, close, output, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

void BullishEngulfing::calculate(const float* input, float* output, int size) noexcept(false) {
    const float* open = input;
    const float* high = input + size;
    const float* low = input + 2 * size;
    const float* close = input + 3 * size;
    calculate(open, high, low, close, output, size);
}
