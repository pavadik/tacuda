#include "hip/hip_runtime.h"
#include <indicators/ROCR.h>
#include <stdexcept>
#include <utils/CudaUtils.h>

__global__ void rocrKernel(const float *__restrict__ input,
                           float *__restrict__ output, int period, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size - period && input[idx] != 0.0f) {
    float prev = input[idx];
    float curr = input[idx + period];
    output[idx] = curr / prev;
  }
}

ROCR::ROCR(int period) : period(period) {}

void ROCR::calculate(const float *input, float *output,
                     int size, hipStream_t stream) noexcept(false) {
  if (period <= 0 || period >= size) {
    throw std::invalid_argument("ROCR: invalid period");
  }
  CUDA_CHECK(hipMemsetAsync(output, 0xFF, size * sizeof(float), stream));
  dim3 block = defaultBlock();
  dim3 grid = defaultGrid(size);
  rocrKernel<<<grid, block, 0, stream>>>(input, output, period, size);
  CUDA_CHECK(hipGetLastError());
}
