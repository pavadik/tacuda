#include "hip/hip_runtime.h"
#include <indicators/VAR.h>
#include <utils/CudaUtils.h>
#include <stdexcept>

__global__ void varKernel(const float* __restrict__ input,
                          float* __restrict__ output,
                          int period, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= period - 1 && idx < size) {
        float mean = 0.0f;
        for (int j = 0; j < period; ++j)
            mean += input[idx - j];
        mean /= period;
        float sumsq = 0.0f;
        for (int j = 0; j < period; ++j) {
            float diff = input[idx - j] - mean;
            sumsq += diff * diff;
        }
        output[idx] = sumsq / period;
    }
}

VAR::VAR(int period) : period(period) {}

void VAR::calculate(const float* input, float* output, int size, hipStream_t stream) noexcept(false) {
    if (period <= 0 || period > size) {
        throw std::invalid_argument("VAR: invalid period");
    }
    CUDA_CHECK(hipMemsetAsync(output, 0xFF, size * sizeof(float), stream));
    dim3 block = defaultBlock();
    dim3 grid = defaultGrid(size);
    varKernel<<<grid, block, 0, stream>>>(input, output, period, size);
    CUDA_CHECK(hipGetLastError());
}
