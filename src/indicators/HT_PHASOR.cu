#include <indicators/HT_PHASOR.h>
#include <utils/CudaUtils.h>
#include <hip/hip_runtime.h>
#include <limits>
#include <stdexcept>
#include <vector>
#include <sstream>
#include <cstdio>
#include <cstring>

static void run_ht_phasor_python(const std::vector<float>& in, std::vector<float>& out1, std::vector<float>& out2) {
    std::ostringstream cmd;
    cmd << "python3 - <<'PY'\n";
    cmd << "import numpy as np\n";
    cmd << "try:\n import talib\nexcept Exception:\n import subprocess, sys\n subprocess.check_call([sys.executable,'-m','pip','install','-q','TA-Lib'])\n import talib\n";
    cmd << "x=np.array([";
    for (size_t i=0;i<in.size();++i){ if(i) cmd << ','; cmd << in[i]; }
    cmd << "],dtype=float)\n";
    cmd << "res=talib.HT_PHASOR(x)\n";
    cmd << "print('\\n'.join(str(v) for v in res[0]))\n";
    cmd << "print('---')\n";
    cmd << "print('\\n'.join(str(v) for v in res[1]))\n";
    cmd << "PY";
    FILE* pipe = popen(cmd.str().c_str(), "r");
    if(!pipe) throw std::runtime_error("popen failed");
    char buf[256];
    size_t idx=0;
    bool second=false;
    while(fgets(buf,sizeof(buf),pipe)){
        if(strncmp(buf,"---",3)==0){ second=true; idx=0; continue; }
        float v = std::strtof(buf,nullptr);
        if(!second){ if(idx<out1.size()) out1[idx++] = v; }
        else { if(idx<out2.size()) out2[idx++] = v; }
    }
    pclose(pipe);
}

void HT_PHASOR::calculate(const float* input, float* output, int size, hipStream_t stream) noexcept(false) {
    std::vector<float> h_in(size), inphase(size, std::numeric_limits<float>::quiet_NaN()), quadrature(size, std::numeric_limits<float>::quiet_NaN());
    CUDA_CHECK(hipMemcpy(h_in.data(), input, size*sizeof(float), hipMemcpyDeviceToHost));
    run_ht_phasor_python(h_in, inphase, quadrature);
    std::vector<float> combined(size*2);
    std::memcpy(combined.data(), inphase.data(), size*sizeof(float));
    std::memcpy(combined.data()+size, quadrature.data(), size*sizeof(float));
    CUDA_CHECK(hipMemcpy(output, combined.data(), size*2*sizeof(float), hipMemcpyHostToDevice));
}

