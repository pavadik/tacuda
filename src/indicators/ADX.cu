#include "hip/hip_runtime.h"
#include <indicators/ADX.h>
#include <utils/CudaUtils.h>
#include <stdexcept>
#include <math.h>

__global__ void adxKernel(const float* __restrict__ high,
                          const float* __restrict__ low,
                          const float* __restrict__ close,
                          float* __restrict__ output,
                          int period, int size) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        float prevHigh = high[0];
        float prevLow = low[0];
        float prevClose = close[0];

        float dmp_s = 0.0f;
        float dmm_s = 0.0f;
        float tr_s = 0.0f;
        float dx_sum = 0.0f;
        float adx = 0.0f;

        for (int i = 1; i < size; ++i) {
            float upMove = high[i] - prevHigh;
            float downMove = prevLow - low[i];
            float dmPlus = (upMove > downMove && upMove > 0.0f) ? upMove : 0.0f;
            float dmMinus = (downMove > upMove && downMove > 0.0f) ? downMove : 0.0f;
            float tr = fmaxf(high[i] - low[i],
                             fmaxf(fabsf(high[i] - prevClose),
                                   fabsf(low[i] - prevClose)));

            prevHigh = high[i];
            prevLow = low[i];
            prevClose = close[i];

            if (i <= period) {
                dmp_s += dmPlus;
                dmm_s += dmMinus;
                tr_s += tr;
                if (i == period) {
                    float dip = (tr_s == 0.0f) ? 0.0f : 100.0f * dmp_s / tr_s;
                    float dim = (tr_s == 0.0f) ? 0.0f : 100.0f * dmm_s / tr_s;
                    float dx = (dip + dim == 0.0f) ? 0.0f :
                               100.0f * fabsf(dip - dim) / (dip + dim);
                    dx_sum = dx;
                }
            } else {
                dmp_s = dmp_s - dmp_s / period + dmPlus;
                dmm_s = dmm_s - dmm_s / period + dmMinus;
                tr_s = tr_s - tr_s / period + tr;
                float dip = (tr_s == 0.0f) ? 0.0f : 100.0f * dmp_s / tr_s;
                float dim = (tr_s == 0.0f) ? 0.0f : 100.0f * dmm_s / tr_s;
                float dx = (dip + dim == 0.0f) ? 0.0f :
                           100.0f * fabsf(dip - dim) / (dip + dim);
                if (i < 2 * period) {
                    dx_sum += dx;
                    if (i == 2 * period - 1) {
                        adx = dx_sum / period;
                        output[i] = adx;
                    }
                } else {
                    adx = (adx * (period - 1) + dx) / period;
                    output[i] = adx;
                }
            }
        }
    }
}

ADX::ADX(int period) : period(period) {}

void ADX::calculate(const float* high, const float* low, const float* close,
                    float* output, int size, hipStream_t stream) noexcept(false) {
    if (period <= 0 || period > size) {
        throw std::invalid_argument("ADX: invalid period");
    }
    CUDA_CHECK(hipMemset(output, 0xFF, size * sizeof(float)));
    adxKernel<<<1, 1, 0, stream>>>(high, low, close, output, period, size);
    CUDA_CHECK(hipGetLastError());
}

void ADX::calculate(const float* input, float* output, int size, hipStream_t stream) noexcept(false) {
    const float* high = input;
    const float* low = input + size;
    const float* close = input + 2 * size;
    calculate(high, low, close, output, size, stream);
}
