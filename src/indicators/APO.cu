#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdexcept>
#include <indicators/APO.h>
#include <utils/CudaUtils.h>

static __device__ float ema_at(const float* __restrict__ x, int idx, int period) {
    const float k = 2.0f / (period + 1.0f);
    float weight = 1.0f;
    float weightedSum = x[idx];
    float weightSum = 1.0f;
    int steps = min(period, idx);
#pragma unroll
    for (int i = 1; i <= steps; ++i) {
        weight *= (1.0f - k);
        weightedSum += x[idx - i] * weight;
        weightSum += weight;
    }
    return weightedSum / weightSum;
}

__global__ void apoKernel(const float* __restrict__ input,
                          float* __restrict__ output,
                          int fastP, int slowP, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= slowP && idx < size) {
        float emaFast = ema_at(input, idx, fastP);
        float emaSlow = ema_at(input, idx, slowP);
        output[idx] = emaFast - emaSlow;
    }
}

tacuda::APO::APO(int fastPeriod, int slowPeriod)
    : fastPeriod(fastPeriod), slowPeriod(slowPeriod) {}

void tacuda::APO::calculate(const float* input, float* output, int size, hipStream_t stream) noexcept(false) {
    if (fastPeriod <= 0 || slowPeriod <= 0) {
        throw std::invalid_argument("APO: invalid periods");
    }
    if (fastPeriod >= slowPeriod) {
        throw std::invalid_argument("APO: fastPeriod must be < slowPeriod");
    }
    CUDA_CHECK(hipMemsetAsync(output, 0xFF, size * sizeof(float), stream));
    dim3 block = defaultBlock();
    dim3 grid = defaultGrid(size);
    apoKernel<<<grid, block, 0, stream>>>(input, output, fastPeriod, slowPeriod, size);
    CUDA_CHECK(hipGetLastError());
}
