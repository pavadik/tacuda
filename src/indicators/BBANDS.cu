#include "hip/hip_runtime.h"
#include <indicators/BBANDS.h>
#include <utils/CudaUtils.h>
#include <utils/DeviceBufferPool.h>
#include <stdexcept>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <cmath>

__global__ void squareKernel(const float* __restrict__ input,
                             float* __restrict__ squared,
                             int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float v = input[idx];
        squared[idx] = v * v;
    }
}

__global__ void bbandsKernel(const float* __restrict__ prefix,
                             const float* __restrict__ prefixSq,
                             float* __restrict__ upper,
                             float* __restrict__ middle,
                             float* __restrict__ lower,
                             int period, int size,
                             float upMult, float downMult) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx <= size - period) {
        float prev = (idx == 0) ? 0.0f : prefix[idx - 1];
        float prevSq = (idx == 0) ? 0.0f : prefixSq[idx - 1];
        float sum = prefix[idx + period - 1] - prev;
        float sumSq = prefixSq[idx + period - 1] - prevSq;
        float mean = sum / period;
        float variance = sumSq / period - mean * mean;
        variance = variance > 0.0f ? variance : 0.0f;
        float stddev = sqrtf(variance);
        middle[idx] = mean;
        upper[idx] = mean + upMult * stddev;
        lower[idx] = mean - downMult * stddev;
    }
}

BBANDS::BBANDS(int period, float upperMultiplier, float lowerMultiplier)
    : period(period), upperMultiplier(upperMultiplier), lowerMultiplier(lowerMultiplier) {}

void BBANDS::calculate(const float* input, float* output, int size, hipStream_t stream) noexcept(false) {
    if (period <= 0 || period > size) {
        throw std::invalid_argument("BBANDS: invalid period");
    }
    // Initialize outputs with NaNs so unwritten tail retains NaN semantics
    CUDA_CHECK(hipMemsetAsync(output, 0xFF, 3 * size * sizeof(float), stream));

    auto prefix = acquireDeviceBuffer<float>(size);
    auto squared = acquireDeviceBuffer<float>(size);
    auto prefixSq = acquireDeviceBuffer<float>(size);

    dim3 block = defaultBlock();
    dim3 grid = defaultGrid(size);
    squareKernel<<<grid, block, 0, stream>>>(input, squared.get(), size);
    CUDA_CHECK(hipGetLastError());

    thrust::device_ptr<const float> inPtr(input);
    thrust::device_ptr<float> prePtr(prefix.get());
    thrust::inclusive_scan(inPtr, inPtr + size, prePtr);

    thrust::device_ptr<float> sqPtr(squared.get());
    thrust::device_ptr<float> preSqPtr(prefixSq.get());
    thrust::inclusive_scan(sqPtr, sqPtr + size, preSqPtr);

    float* upper = output;
    float* middle = output + size;
    float* lower = output + 2 * size;
    bbandsKernel<<<grid, block, 0, stream>>>(prefix.get(), prefixSq.get(), upper, middle, lower,
                                  period, size, upperMultiplier, lowerMultiplier);
    CUDA_CHECK(hipGetLastError());
}

