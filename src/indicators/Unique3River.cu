#include "hip/hip_runtime.h"
#include <indicators/Unique3River.h>
#include <utils/CandleUtils.h>
#include <utils/CudaUtils.h>

__global__ void unique3RiverKernel(const float* __restrict__ open,
                                   const float* __restrict__ high,
                                   const float* __restrict__ low,
                                   const float* __restrict__ close,
                                   float* __restrict__ output, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx > 1 && idx < size) {
    output[idx] =
        is_unique_3_river(open[idx - 2], high[idx - 2], low[idx - 2],
                          close[idx - 2], open[idx - 1], high[idx - 1],
                          low[idx - 1], close[idx - 1], open[idx], high[idx],
                          low[idx], close[idx])
            ? 1.0f
            : 0.0f;
  }
}

void Unique3River::calculate(const float* open, const float* high,
                              const float* low, const float* close,
                              float* output, int size) noexcept(false) {
  CUDA_CHECK(hipMemset(output, 0xFF, size * sizeof(float)));
  dim3 block = defaultBlock();
  dim3 grid = defaultGrid(size);
  unique3RiverKernel<<<grid, block>>>(open, high, low, close, output, size);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}

void Unique3River::calculate(const float* input, float* output,
                              int size) noexcept(false) {
  const float* open = input;
  const float* high = input + size;
  const float* low = input + 2 * size;
  const float* close = input + 3 * size;
  calculate(open, high, low, close, output, size);
}

