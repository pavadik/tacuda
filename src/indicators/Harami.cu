#include "hip/hip_runtime.h"
#include <indicators/Harami.h>
#include <utils/CandleUtils.h>
#include <utils/CudaUtils.h>

__global__ void haramiKernel(const float* __restrict__ open,
                             const float* __restrict__ high,
                             const float* __restrict__ low,
                             const float* __restrict__ close,
                             float* __restrict__ output,
                             int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > 0 && idx < size) {
        output[idx] = harami(open[idx - 1], close[idx - 1], open[idx], close[idx]);
    }
}

void Harami::calculate(const float* open, const float* high,
                       const float* low, const float* close,
                       float* output, int size, hipStream_t stream) noexcept(false) {
    CUDA_CHECK(hipMemset(output, 0xFF, size * sizeof(float)));
    dim3 block = defaultBlock();
    dim3 grid = defaultGrid(size);
    haramiKernel<<<grid, block, 0, stream>>>(open, high, low, close, output, size);
    CUDA_CHECK(hipGetLastError());
}

void Harami::calculate(const float* input, float* output,
                       int size, hipStream_t stream) noexcept(false) {
    const float* open = input;
    const float* high = input + size;
    const float* low  = input + 2 * size;
    const float* close= input + 3 * size;
    calculate(open, high, low, close, output, size, stream);
}

