#include "hip/hip_runtime.h"
#include <indicators/ADXR.h>
#include <indicators/ADX.h>
#include <utils/CudaUtils.h>
#include <utils/DeviceBufferPool.h>
#include <stdexcept>

__global__ void adxrKernel(const float* __restrict__ adx,
                           float* __restrict__ output,
                           int period, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int start = 3 * period - 1;
    if (idx >= start && idx < size) {
        output[idx] = 0.5f * (adx[idx] + adx[idx - period]);
    }
}

tacuda::ADXR::ADXR(int period) : period(period) {}

void tacuda::ADXR::calculate(const float* high, const float* low, const float* close,
                     float* output, int size, hipStream_t stream) noexcept(false) {
    if (period <= 0 || period > size) {
        throw std::invalid_argument("ADXR: invalid period");
    }
    auto adx = acquireDeviceBuffer<float>(size);

    tacuda::ADX adxInd(period);
    adxInd.calculate(high, low, close, adx.get(), size, stream);

    CUDA_CHECK(hipMemsetAsync(output, 0xFF, size * sizeof(float), stream));
    dim3 block = defaultBlock();
    dim3 grid = defaultGrid(size);
    adxrKernel<<<grid, block, 0, stream>>>(adx.get(), output, period, size);
    CUDA_CHECK(hipGetLastError());
}

void tacuda::ADXR::calculate(const float* input, float* output, int size, hipStream_t stream) noexcept(false) {
    const float* high = input;
    const float* low = input + size;
    const float* close = input + 2 * size;
    calculate(high, low, close, output, size, stream);
}
