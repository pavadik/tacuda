#include "hip/hip_runtime.h"
#include <indicators/HomingPigeon.h>
#include <utils/CandleUtils.h>
#include <utils/CudaUtils.h>

__global__ void homingPigeonKernel(const float* __restrict__ open,
                                   const float* __restrict__ high,
                                   const float* __restrict__ low,
                                   const float* __restrict__ close,
                                   float* __restrict__ output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > 0 && idx < size) {
        output[idx] = is_homing_pigeon(open[idx - 1], high[idx - 1],
                                       low[idx - 1], close[idx - 1], open[idx],
                                       high[idx], low[idx], close[idx])
                          ? 1.0f
                          : 0.0f;
    }
}

void HomingPigeon::calculate(const float* open, const float* high,
                             const float* low, const float* close,
                             float* output, int size, hipStream_t stream) noexcept(false) {
    CUDA_CHECK(hipMemsetAsync(output, 0xFF, size * sizeof(float), stream));
    dim3 block = defaultBlock();
    dim3 grid = defaultGrid(size);
    homingPigeonKernel<<<grid, block, 0, stream>>>(open, high, low, close, output, size);
    CUDA_CHECK(hipGetLastError());
}

void HomingPigeon::calculate(const float* input, float* output,
                             int size, hipStream_t stream) noexcept(false) {
    const float* open = input;
    const float* high = input + size;
    const float* low  = input + 2 * size;
    const float* close= input + 3 * size;
    calculate(open, high, low, close, output, size, stream);
}

