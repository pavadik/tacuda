#include "hip/hip_runtime.h"
#include <indicators/NVI.h>
#include <utils/CudaUtils.h>
#include <stdexcept>

namespace {
__global__ void nviKernel(const float* __restrict__ close,
                          const float* __restrict__ volume,
                          float* __restrict__ output,
                          int size) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        if (size <= 0) {
            return;
        }
        float prevIndex = 1000.0f;
        output[0] = prevIndex;
        for (int i = 1; i < size; ++i) {
            float prevVol = volume[i - 1];
            float currVol = volume[i];
            float prevClose = close[i - 1];
            float value = prevIndex;
            if (currVol < prevVol && prevClose != 0.0f) {
                value = prevIndex * (1.0f + (close[i] - prevClose) / prevClose);
            }
            output[i] = value;
            prevIndex = value;
        }
    }
}
} // namespace

void tacuda::NVI::calculate(const float* close, const float* volume, float* output,
                            int size, hipStream_t stream) noexcept(false) {
    if (size <= 0) {
        throw std::invalid_argument("NVI: invalid size");
    }
    CUDA_CHECK(hipMemsetAsync(output, 0xFF, size * sizeof(float), stream));
    nviKernel<<<1, 1, 0, stream>>>(close, volume, output, size);
    CUDA_CHECK(hipGetLastError());
}

void tacuda::NVI::calculate(const float* input, float* output, int size,
                            hipStream_t stream) noexcept(false) {
    const float* close = input;
    const float* volume = input + size;
    calculate(close, volume, output, size, stream);
}
