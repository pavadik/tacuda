#include "hip/hip_runtime.h"
#include <indicators/LINEARREG_SLOPE.h>
#include <utils/CudaUtils.h>
#include <stdexcept>
#include <math.h>

__global__ void linearregSlopeKernel(const float* __restrict__ in,
                                     float* __restrict__ out,
                                     int period, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx <= size - period) {
        float sumY = 0.0f, sumXY = 0.0f;
        for (int i = 0; i < period; ++i) {
            float y = in[idx + i];
            sumY += y;
            sumXY += i * y;
        }
        float sumX = 0.5f * period * (period - 1);
        float sumX2 = (period - 1) * period * (2 * period - 1) / 6.0f;
        float denom = period * sumX2 - sumX * sumX;
        float slope = (period * sumXY - sumX * sumY) / denom;
        out[idx] = slope;
    }
}

LINEARREG_SLOPE::LINEARREG_SLOPE(int period) : period(period) {}

void LINEARREG_SLOPE::calculate(const float* input, float* output, int size, hipStream_t stream) noexcept(false) {
    if (period <= 0 || period > size) {
        throw std::invalid_argument("LINEARREG_SLOPE: invalid period");
    }
    CUDA_CHECK(hipMemsetAsync(output, 0xFF, size * sizeof(float), stream));
    dim3 block = defaultBlock();
    dim3 grid = defaultGrid(size);
    linearregSlopeKernel<<<grid, block, 0, stream>>>(input, output, period, size);
    CUDA_CHECK(hipGetLastError());
}

