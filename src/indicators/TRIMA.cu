#include <indicators/SMA.h>
#include <indicators/TRIMA.h>
#include <stdexcept>
#include <utils/CudaUtils.h>

TRIMA::TRIMA(int period) : period(period) {}

void TRIMA::calculate(const float *input, float *output,
                      int size) noexcept(false) {
  if (period <= 0 || size < period) {
    throw std::invalid_argument("TRIMA: invalid period");
  }
  CUDA_CHECK(hipMemset(output, 0xFF, size * sizeof(float)));
  int p1 = (period + 1) / 2;
  int p2 = (period % 2 == 0) ? (p1 + 1) : p1;

  float *tmp = nullptr;
  CUDA_CHECK(hipMalloc(&tmp, size * sizeof(float)));

  SMA sma1(p1);
  sma1.calculate(input, tmp, size);
  int size2 = size - p1 + 1;
  SMA sma2(p2);
  sma2.calculate(tmp, output, size2);

  CUDA_CHECK(hipFree(tmp));
}
