#include <indicators/SMA.h>
#include <indicators/TRIMA.h>
#include <stdexcept>
#include <utils/CudaUtils.h>
#include <utils/DeviceBufferPool.h>

tacuda::TRIMA::TRIMA(int period) : period(period) {}

void tacuda::TRIMA::calculate(const float *input, float *output,
                      int size, hipStream_t stream) noexcept(false) {
  if (period <= 0 || size < period) {
    throw std::invalid_argument("TRIMA: invalid period");
  }
  CUDA_CHECK(hipMemsetAsync(output, 0xFF, size * sizeof(float), stream));
  int p1 = (period + 1) / 2;
  int p2 = (period % 2 == 0) ? (p1 + 1) : p1;

  auto tmp = acquireDeviceBuffer<float>(size);

  tacuda::SMA sma1(p1);
  sma1.calculate(input, tmp.get(), size, stream);
  int size2 = size - p1 + 1;
  tacuda::SMA sma2(p2);
  sma2.calculate(tmp.get(), output, size2);
}
