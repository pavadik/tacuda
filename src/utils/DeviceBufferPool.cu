#include <utils/DeviceBufferPool.h>
#include <utils/CudaUtils.h>

DeviceBufferPool& DeviceBufferPool::instance() {
    static DeviceBufferPool pool;
    return pool;
}

void* DeviceBufferPool::acquire(size_t bytes) {
    if (bytes == 0) {
        return nullptr;
    }
    std::lock_guard<std::mutex> lock(mutex);
    auto& vec = freeBuffers[bytes];
    if (!vec.empty()) {
        void* ptr = vec.back();
        vec.pop_back();
        return ptr;
    }
    void* ptr = nullptr;
    CUDA_CHECK(hipMalloc(&ptr, bytes));
    sizes[ptr] = bytes;
    ++allocations;
    return ptr;
}

void DeviceBufferPool::release(void* ptr) {
    if (!ptr) return;
    std::lock_guard<std::mutex> lock(mutex);
    auto it = sizes.find(ptr);
    if (it != sizes.end()) {
        freeBuffers[it->second].push_back(ptr);
    }
}

void DeviceBufferPool::cleanup() {
    std::lock_guard<std::mutex> lock(mutex);
    for (auto &kv : sizes) {
        hipFree(kv.first);
    }
    freeBuffers.clear();
    sizes.clear();
    allocations = 0;
}

DeviceBufferPool::~DeviceBufferPool() { cleanup(); }

size_t DeviceBufferPool::allocationCount() const {
    std::lock_guard<std::mutex> lock(mutex);
    return allocations;
}
